
#include <hip/hip_runtime.h>
#include <stdio.h>

#include <stdlib.h>

#include <time.h>



#define BLOCK_SIZE 16



// Function to initialize the Ising model with a random initial state

void initializeIsingModel(int size, char *isingModel) {

    for (int i = 0; i < size * size; ++i) {

        isingModel[i] = (rand() % 2) ? '*' : ' ';  // Randomly assign '*' or ' '

    }

}



// Function to print the Ising model

void printIsingModel(int size, char *isingModel) {

    for (int i = 0; i < size; ++i) {

        for (int j = 0; j < size; ++j) {

            printf("%c ", isingModel[i * size + j]);

        }

        printf("\n");

    }

    printf("\n");

}



// Kernel function for GPU parallelization with each thread computing a block of moments

__global__ void updateIsingModelGPUV2(int size, char *isingModel) {

    int i = blockIdx.x * blockDim.x + threadIdx.x;

    int j = blockIdx.y * blockDim.y + threadIdx.y;



    // Compute a block of moments

    for (int i_local = i; i_local < i + blockDim.x * 2; i_local += blockDim.x) {

        for (int j_local = j; j_local < j + blockDim.y * 2; j_local += blockDim.y) {

            if (i_local < size && j_local < size) {

                int sum = (isingModel[((i_local - 1 + size) % size) * size + j_local] == '*') +

                          (isingModel[i_local * size + (j_local - 1 + size) % size] == '*') +

                          (isingModel[i_local * size + j_local] == '*') +

                          (isingModel[((i_local + 1) % size) * size + j_local] == '*') +

                          (isingModel[i_local * size + (j_local + 1) % size] == '*');



                isingModel[i_local * size + j_local] = (sum > 2) ? '*' : ' ';

            }

        }

    }

}



// Function to copy Ising model from device to host

void copyModelFromDevice(int size, char *hostModel, char *deviceModel) {

    hipMemcpy(hostModel, deviceModel, size * size * sizeof(char), hipMemcpyDeviceToHost);

}



int main() {

    srand(time(NULL));  // Seed for random number generation



    int size = 5;       // Size of the 2D lattice

    int iterations = 3; // Number of iterations



    char *hostModel = (char *)malloc(size * size * sizeof(char));

    char *deviceModel;



    // Initialize the Ising model with a random initial state

    initializeIsingModel(size, hostModel);



    // Print the initial state

    printf("Initial Ising Model:\n");

    printIsingModel(size, hostModel);



    // Allocate memory on the GPU for Ising model

    hipMalloc((void **)&deviceModel, size * size * sizeof(char));



    // Copy the initial Ising model from host to device

    hipMemcpy(deviceModel, hostModel, size * size * sizeof(char), hipMemcpyHostToDevice);



    // Define GPU grid and block dimensions

    dim3 blockDim(BLOCK_SIZE, BLOCK_SIZE);

    dim3 gridDim((size + blockDim.x * 2 - 1) / (blockDim.x * 2), (size + blockDim.y * 2 - 1) / (blockDim.y * 2));



    // Perform iterations of the Ising model simulation on GPU

    for (int k = 0; k < iterations; ++k) {

        updateIsingModelGPUV2<<<gridDim, blockDim>>>(size, deviceModel);

        hipDeviceSynchronize();  // Wait for GPU to finish



        // Copy the updated Ising model from device to host

        copyModelFromDevice(size, hostModel, deviceModel);



        printf("After Iteration %d:\n", k + 1);

        printIsingModel(size, hostModel);

    }



    // Free allocated memory on the GPU

    hipFree(deviceModel);



    free(hostModel);



    return 0;

}


